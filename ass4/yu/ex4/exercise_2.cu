#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define ARRAY_SIZE 10000
#define TPB 256

inline int GET_BLOCKS(const int n, const int tpb)
{
  return (n + tpb - 1) / tpb;
}

inline int GET_THREADS(const int n, const int tpb)
{
	if(n>=tpb){
		return tpb;
	}else{
		return tpb;
	}
}


__global__ void saxpy_device (int n, float* ptr_x, float* ptr_y, float a)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)
	{
		ptr_y[i] += a * ptr_x[i];
	}	
}

void saxpy_host(int n, float* ptr_x, float* ptr_y, float a)
{
	for (int i=0; i<n; i++){
		ptr_y[i] += a * ptr_x[i];
	}
}

int main()
{
	float a = 2.5;

        float *x_host = (float*)malloc(ARRAY_SIZE * sizeof(float));
        float *y_host = (float*)malloc(ARRAY_SIZE * sizeof(float));
        float *y_device = (float*)malloc(ARRAY_SIZE * sizeof(float));
        float *x_gpu, *y_gpu;
	hipMalloc((int**)(&x_gpu), ARRAY_SIZE * sizeof(float));
        hipMalloc((int**)(&y_gpu), ARRAY_SIZE * sizeof(float));

        srand(time(NULL));
        for(int i=0; i<ARRAY_SIZE; i++)  
        {
                float x = ((float)rand()/(float)(RAND_MAX)) * 5.0;
                float y = ((float)rand()/(float)(RAND_MAX)) * 5.0;
                x_host[i] = x;
                y_host[i] = y;
                y_device[i] = y;
        }

        hipMemcpy(x_gpu, x_host, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(y_gpu, y_device, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	
	printf("Computing SAXPY on CPU ... "); 
        saxpy_host(ARRAY_SIZE, x_host, y_host, a);
	printf("Done!\n");

	printf("Computing SAXPY on GPU ... ");
        saxpy_device<<< GET_BLOCKS(ARRAY_SIZE, TPB), GET_THREADS(ARRAY_SIZE, TPB) >>>(ARRAY_SIZE, x_gpu, y_gpu, a);
        hipMemcpy(y_device, y_gpu, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	printf("Done!\n");
	
	printf("Compare the output for each implementation ... ");
	bool success = true;
        for(int i=0; i<ARRAY_SIZE; i++)
        {
                if(abs(y_host[i]-y_device[i]) > 0.00001)
                {
                        printf("Error at position %d! %f != %f\n", i, y_host[i], y_device[i]);
         		success = false;
			break;
                }
        }
	if(success){
		printf("Correct!");
	}

        free(x_host);
        free(y_host);
        free(y_device);
        hipFree(x_gpu);
        hipFree(y_gpu);
        
        return 0;
}
