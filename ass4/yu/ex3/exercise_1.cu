
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256
#define TPB 128

__global__ void print()
{
  const int i = blockIdx.x*blockDim.x + threadIdx.x;
  printf("Hello World! My threadId is %d\n", i);
}

int main()
{
  print<<<N/TPB, TPB>>>();
  hipDeviceSynchronize();
  return 0;
}
