
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 256
#define TPB 256

__global__ void print()
{
  int t_ind = blockIdx.x*blockDim.x + threadIdx.x;
  printf("Hello World! My threadId is %d from block %d\n", t_ind, blockIdx.x);
}

int main()
{
  print<<<N/TPB, TPB>>>();
  hipDeviceSynchronize();
  return 0;
}
