#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#define TPB 256
#define ARRAY_SIZE 1000000000
#define A 3.14

__global__ void saxpy(float* x, float* y, float a) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < ARRAY_SIZE) {
		y[i] += a*x[i];
	}
}

void saxpy_cpu(float* x, float* y, float a) {
	for (int i = 0; i < ARRAY_SIZE; i++) {
		y[i] += x[i]*a;
	}
}

double elapsed(timeval t1, timeval t2) {
	return (1000000*(t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec)/1000.0;
}


int main() {
	float* x, *y, *x_gpu, *y_gpu, *x_cpu, *y_cpu;
	float a = A;
	struct timeval t1, t2;

	x = (float*)malloc(sizeof(float)*ARRAY_SIZE);
	y = (float*)malloc(sizeof(float)*ARRAY_SIZE);
	x_cpu = (float*)malloc(sizeof(float)*ARRAY_SIZE);
	y_cpu = (float*)malloc(sizeof(float)*ARRAY_SIZE);

	srand(time(NULL));
	for (int i = 0; i < ARRAY_SIZE; i++) {
		x[i] = (float)rand();
		x_cpu[i] = x[i];
		y[i] = (float)rand();
		y_cpu[i] = y[i];
	}

	hipMalloc(&x_gpu, sizeof(float)*ARRAY_SIZE);
	hipMalloc(&y_gpu, sizeof(float)*ARRAY_SIZE);
	
	gettimeofday(&t1, NULL);
	hipMemcpy(x_gpu, x, sizeof(float)*ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(float)*ARRAY_SIZE, hipMemcpyHostToDevice);

	saxpy<<<(ARRAY_SIZE + TPB - 1)/TPB, TPB>>>(x_gpu, y_gpu, a);	
	hipDeviceSynchronize();
	
	hipMemcpy(y, y_gpu, sizeof(float)*ARRAY_SIZE, hipMemcpyDeviceToHost);
	gettimeofday(&t2, NULL);
	
	printf("GPU calculations complete! Took %f ms\n\n", elapsed(t1,t2));

	gettimeofday(&t1, NULL);
	saxpy_cpu(x_cpu, y_cpu, a);	
	gettimeofday(&t2, NULL);
	printf("CPU calculations complete! Took %f ms\n\n", elapsed(t1, t2));

	bool same = true;
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (abs(y_cpu[i] - y[i])/y[i] > 0.0001) {
			printf("Arrays differ! %f != %f\n", y_cpu[i], y[i]);
			same = false;
			break;
		}
	}
	if (same) {
		printf("Result check is complete! Arrays the same!\n");
	}

	free(x);
	free(y);
	free(x_cpu);
	free(y_cpu);
	hipFree(x_gpu);
	hipFree(y_gpu);

	return 0;
}
